#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "../shader.cu"

#define EPS 0.002
#define MIN_DIST 0.01
__device__ vec2 Sky(vec3 p){
  return vec2(abs(dist(p,vec3(0))-25)-0.2,4);
}

__device__ vec2 map(vec3 p){  
  // mat 4 is for mirroring surfaces
  vec2 plane = vec2(abs(p.y +1 + 0.015*sin(frame/15+length(vec2(p.x,p.z))*3)),5); 
  vec2 mirrorball = vec2(dist(p+vec3(0.7-3-3*sinfr,-3.8-sinfr,-3.2+cosfr),vec3(0.5))-3,4);
  vec2 mirrorcube = vec2(length(max(abs(p-vec3(12,3+sinfr,12))-vec3(2.5),0.0/*+sinfr*0.1275*/))-0.225,4);
  //p.x = fract(p.x/4)*4-2;
  //mat 1 is a greybluish surface
  vec2 ball = vec2(dist(p-vec3(0,1,0),vec3(0.5))-1,15);
  vec2 cube = vec2(length(max(abs(p-vec3(-2,1+sinfr,2))-vec3(0.5),0.0+sinfr*0.275))-0.5,15);
  return min(min(min(min(min(mirrorball,ball),plane),Sky(p)),cube),mirrorcube);
}

__device__ vec3 normal(vec3 p){
  vec3 q = vec3(map(vec3(p.x + EPS, p.y, p.z)).x - map(vec3(p.x - EPS, p.y, p.z)).x,
            map(vec3(p.x, p.y + EPS, p.z )).x - map(vec3(p.x, p.y - EPS, p.z)).x,
            map(vec3(p.x, p.y, p.z + EPS)).x - map(vec3(p.x, p.y, p.z - EPS)).x);
  return normalize(q); 
} 
 
__device__ vec3 trace(vec3 org, vec3 dir){
   vec3 cl,p;
   vec3 light = vec3(sinfr*2,5.0+3*sinfr,-2.0);
   vec2 d;
   float dist = 0.0;
   float off_dist = 0;   
   for(int i = 0; i < 1024; i++)
   {
     //d.y = 0;
     p = org+dir*dist;
     d = map(p);
     if( d.x <= MIN_DIST){
          if (d.y==4){
            off_dist += dist;
            dir = reflect(dir,normal(p));
            dist = 1.1*MIN_DIST;
            org = p;         
          }
          else if(d.y== 15){//glass
            off_dist += dist;
            org = p;
            dir = dir + normal(p)+vec3(sinfr);
            dist = 1.1*MIN_DIST;
/*            while(d.y == 15)
            {
              p = org+dir*dist;
              d = map(p);
              dist += d.x;
            }
*/            
          }
          else
            break;  
     }
     else
       d.y = 0;
     dist += d.x;
   }
   dist-=5*off_dist;
    
   vec3 norm = normal(p);
   vec3 reflection =  dir - norm* 2 * dot(dir, norm);
   vec3 c3po = vec3(0.8,1.0,0.8);
   c3po = c3po * dot(norm, normalize(light-p));
   vec3 ambient = vec3(0.3,0.4,0.65);
   c3po = c3po + ambient + vec3(1,1,1);
   float spec = pow(max(0.0,dot(reflection,normalize(light-p))),10);
   cl.x = dist*15*norm.x;
   cl.y = dist*15*norm.y;
   cl.z = dist*15*norm.z;
   if(d.y==5)//sky
     cl = (c3po+ vec3(1)*spec+ambient)*35;//(c3po+  vec3(1)*spec+ambient)*40;
   else if(d.y== 2 || d.y == 1) //ball
   {
       cl = (c3po+  vec3(1)*spec+ambient)*50;
   }
   else if(d.y == 0)
     cl = vec3(0);
   if(d.y == 2) //cube
     {cl.z = cl.y/5;
      //cl.x *= cl.z;
     }
   if (d.y == 1){
     cl.z *= cl.y/4;}
   if(d.y == 10){
     cl = vec3(50,50*abs(p.x*p.y),255);
     cl.y -= off_dist;
     }
   if (d.y == 5)
   {
     cl.x /= 5;
     if (off_dist > 0)
       cl.y-= off_dist/2; 
   }
   return cl;
}

__global__ void Mandel_calc(unsigned char* image_buffer){
  unsigned short int row = (blockIdx.y * blockDim.y + threadIdx.y);  // WIDTH
  unsigned short int col = (blockIdx.x * blockDim.x + threadIdx.x);  // HEIGHT
  unsigned int idx = 3*(row * window.x + col);

  float y0 = - (float) (row -window.x/2)/(window.x/2)/2;
  float x0 = (float) (col -window.y/2)/(window.y/2)/2;
   
  vec3 direction = normalize(vec3(x0+0.5+sinfr*0.3,y0-0.6, 1.0));
  vec3 origin =  vec3(1.0-3,7.0,-12.0 );
  vec3 cl = trace(origin,direction);
  color(cl,&image_buffer[idx]);
  }



extern "C" {
 unsigned char*  d_image_buffer;
 unsigned int arr_size;

__host__ void init_cuda(const int width, const int height){
  arr_size = 3 * width * height;
  hipMallocManaged(&d_image_buffer, arr_size*sizeof(unsigned char));
  printf("Cuda Memory allocated\n");
}

__host__ void Mandel(const int width, const int height,unsigned char* image_buffer){
  dim3 block_size(16, 16);
  dim3 grid_size(width / block_size.x, height / block_size.y);
  Mandel_calc<<<grid_size, block_size>>>(d_image_buffer);
  hipPeekAtLastError();
  hipDeviceSynchronize();
  hipMemcpy(image_buffer, d_image_buffer, arr_size, hipMemcpyDeviceToHost);
  }
__host__ void exit_cuda(){
  hipFree(d_image_buffer);
  printf("CudaMemory free\n");
  
}

}

#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "../shader.cu"
#define EPS 0.001

__device__ float trace(vec3 org, vec3 dir);

__device__ float MyBall(vec3 p,vec3 pos, float r){
  return dist(p,pos)-r;
}

__device__ float mandelbulb(vec3 pos){
  float dr = 1;
  float r = 1;
  vec3 z = pos;
  float power =3 + cosfr;
  for(int i = 0; i < 15; i++){
    r = length(z);
    if(r>2.5)
      break;
    float theta = acos(z.z /r)  * power;
    float phi = atan2(z.y,z.x) * power;
    float zr = pow(r,power);
    dr = pow(r,power-1)*power*dr+1;
    
    z = vec3(sin(theta)*cos(phi),sin(phi)*sin(theta),cos(theta))*zr;
    z = z+ pos;
  
  }
  return 0.5 * log(r) * r /dr;
}

__device__ float Skull(vec3 p){
  float plane = 12.5-p.z+0.3*sin(frame/15 + length(vec2(p.x,p.y)));

  float ball = abs(MyBall(p,vec3(0,2 ,0),2))-0.025;
  float eye1 = abs(MyBall(p,vec3(-0.2,2.5,-1.7),0.5+sinfr*0.06))-0.05+sinfr*0.06;
  float eye2 = abs(MyBall(p,vec3(1.0,2.5,-1.7),0.5))-0.025+cosfr*0.03;
  
  float cube = abs(length(max(abs(p-vec3(0.1,0.4+0.2*sinfr,-0.2))-vec3(1,0.4,1),0))-0.2)-0.05;
  return min(smin(max(max(ball,-eye1),-eye2),cube,0.5),plane);
}

__device__ float map(vec3 p){ 
  //return mandelbulb(p);
  return min(Skull(p),mandelbulb(p));
}

__device__ vec3 normal(vec3 p){
  vec3 q = vec3(map(vec3(p.x + EPS, p.y, p.z)) - map(vec3(p.x - EPS, p.y, p.z)),
            map(vec3(p.x, p.y + EPS, p.z )) - map(vec3(p.x, p.y - EPS, p.z)),
            map(vec3(p.x, p.y, p.z + EPS)) - map(vec3(p.x, p.y, p.z - EPS)));
  return normalize(q); 
} 
 
__device__ float trace(vec3 org, vec3 dir){
   float dist = 0.0;   
   float d;
   for(int i = 0; i < 240; i++)
   {
     vec3 p = org+dir*dist;
     d = map(p);
     if( d <= 0.000001){
        break;  
     }
     dist += d;
   }
   return dist;
}

__global__ void Mandel_calc(unsigned char* image_buffer){
  unsigned short int row = (blockIdx.y * blockDim.y + threadIdx.y);  // WIDTH
  unsigned short int col = (blockIdx.x * blockDim.x + threadIdx.x);  // HEIGHT
  unsigned int idx = 3*(row * window.x + col);

  float y0 = - (float) (row -window.x/2)/(window.x/2)*1.5;
  float x0 = (float) (col -window.y/2)/(window.y/2)*1.5;
  float r,g,b;  
  
  
  vec3 direction = normalize(vec3(x0,y0, 1));
  vec3 light = vec3(sinfr*2,5.0+3*cosfr,-2.0);
  vec3 origin =  vec3(1,2,-4);

  float dist = trace(origin,direction);
  vec3 p = origin + direction*dist;
  vec3 norm = normal(p);
  //double f = dot(direction, norm);
  vec3 reflection =  direction - norm* 2 * dot(direction, norm);
  vec3 c3po = vec3(0.8,1.0,0.8);
  c3po = c3po * dot(norm, normalize(light-p));
  float spec = pow(max(0.0,dot(reflection,normalize(light-p))),15);
  vec3 ambient = vec3(0.1,0.1,0.75);
  c3po = c3po + ambient + vec3(1,1,1);

  r = c3po.x*100;
  g = c3po.y*100;
  b = c3po.z*100;
  color(r,g,b,&image_buffer[idx]);
  }


extern "C" {
 unsigned char*  d_image_buffer;
 unsigned int arr_size;

__host__ void init_cuda(const int width, const int height){
  arr_size = 3 * width * height;
  hipMallocManaged(&d_image_buffer, arr_size*sizeof(unsigned char));
  printf("Cuda Memory allocated\n");
}

__host__ void Mandel(const int width, const int height,unsigned char* image_buffer){
  dim3 block_size(16, 16);
  dim3 grid_size(width / block_size.x, height / block_size.y);
  Mandel_calc<<<grid_size, block_size>>>(d_image_buffer);
  hipPeekAtLastError();
  hipDeviceSynchronize();
  hipMemcpy(image_buffer, d_image_buffer, arr_size, hipMemcpyDeviceToHost);
  }
__host__ void exit_cuda(){
  hipFree(d_image_buffer);
  printf("CudaMemory free\n");
  
}

}

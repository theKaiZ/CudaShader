#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "../shader.cu"
#define EPS 0.0075

__device__ float trace(vec3 org, vec3 dir);

__device__ float mandelbulb2(vec3 pos){
  float dr = 1;
  float r = 1;
  vec3 z = pos;
  float power =4 + sinfr;
  for(int i = 0; i < 15; i++){
    r = length(z);
    if(r>2.5)
      break;
    float theta = acos(z.z /r)  * power;
    float phi = atan2(z.y,z.x) * power;
    float zr = pow(r,power);
    dr = pow(r,power-1)*power*dr+1;
    
    z = vec3(sin(theta)*cos(phi),sin(phi)*cos(theta),cos(theta))*zr;
    z = z+ pos;
  
  }
  return 0.5 * log(r) * r /dr;
}

__device__ float mandelbulb(vec3 pos){
  float dr = 1;
  float r = 1;
  vec3 z = pos;
  float power =3 + cosfr;
  for(int i = 0; i < 15; i++){
    r = length(z);
    if(r>2.5)
      break;
    float theta = acos(z.z /r)  * power;
    float phi = atan2(z.y,z.x) * power;
    float zr = pow(r,power);
    dr = pow(r,power-1)*power*dr+1;
    
    z = vec3(sin(theta)*cos(phi),sin(phi)*sin(theta),cos(theta))*zr;
    z = z+ pos;
  
  }
  return 0.5 * log(r) * r /dr;
}

__device__ float map(vec3 p){ 
  float plane = sinfr*4.5 + 5.5-p.z;
  p.y = fract(p.y/3)*3-1.5;
  float bulb1 = abs(mandelbulb(p));
  float bulb2 = abs(mandelbulb2(p));
  return smin(smin(bulb1,bulb2,0.7+0.45*sinfr),plane,2);
}

__device__ vec3 normal(vec3 p){
  vec3 q = vec3(map(vec3(p.x + EPS, p.y, p.z)) - map(vec3(p.x - EPS, p.y, p.z)),
            map(vec3(p.x, p.y + EPS, p.z )) - map(vec3(p.x, p.y - EPS, p.z)),
            map(vec3(p.x, p.y, p.z + EPS)) - map(vec3(p.x, p.y, p.z - EPS)));
  return normalize(q); 
} 
 
__device__ float trace(vec3 org, vec3 dir){
   float dist = 0.0;   
   float d;
   for(int i = 0; i < 140; i++)
   {
     vec3 p = org+dir*dist;
     d = map(p);
     if( d <= 0.00000001){
        break;  
     }
     dist += d;
   }
   return dist;
}

__global__ void Mandel_calc(unsigned char* image_buffer){
  unsigned short int row = (blockIdx.y * blockDim.y + threadIdx.y);  // WIDTH
  unsigned short int col = (blockIdx.x * blockDim.x + threadIdx.x);  // HEIGHT
  unsigned int idx = 3*(row * window.x + col);

  float y0 = - (float) (row -window.x/2)/(window.x/2)*5;
  float x0 = (float) (col -window.y/2)/(window.y/2)*10;
  float r,g,b;  
  
  
  vec3 direction = normalize(vec3(x0,y0, 1));
  vec3 light = vec3(sinfr*2,5.0+7*cosfr,5.0);
  vec3 origin =  vec3(0,0,-2.5);

  float dist = trace(origin,direction);
  vec3 p = origin + direction*dist;
  vec3 norm = normal(p);
  //double f = dot(direction, norm);
  vec3 reflection =  direction - norm* 2 * dot(direction, norm);
  vec3 c3po = vec3(0.8,1.0,0.8);
  c3po = c3po * dot(norm, normalize(light-p));
  float spec = pow(max(0.0,dot(reflection,normalize(light-p))),15);
  vec3 ambient = vec3(0.1,0.1,0.75);
  c3po = c3po + ambient + vec3(1,1,1);

  r = c3po.x*100;
  g = c3po.y*100;
  b = c3po.z*100;
  color(r,g,b,&image_buffer[idx]);
  }


extern "C" {
 unsigned char*  d_image_buffer;
 unsigned int arr_size;

__host__ void init_cuda(const int width, const int height){
  arr_size = 3 * width * height;
  hipMallocManaged(&d_image_buffer, arr_size*sizeof(unsigned char));
  printf("Cuda Memory allocated\n");
}

__host__ void Mandel(const int width, const int height,unsigned char* image_buffer){
  dim3 block_size(16, 16);
  dim3 grid_size(width / block_size.x, height / block_size.y);
  Mandel_calc<<<grid_size, block_size>>>(d_image_buffer);
  hipPeekAtLastError();
  hipDeviceSynchronize();
  hipMemcpy(image_buffer, d_image_buffer, arr_size, hipMemcpyDeviceToHost);
  }
__host__ void exit_cuda(){
  hipFree(d_image_buffer);
  printf("CudaMemory free\n");
  
}

}
